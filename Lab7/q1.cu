#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>

__global__ void countWordOccurrences(char *sentence, char *word, int *count) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int wordLen = strlen(word);
    int sentLen = strlen(sentence);

    if (idx <= sentLen - wordLen && strncmp(&sentence[idx], word, wordLen) == 0) {
        atomicAdd(count, 1);
    }
}

int main() {
    char sentence[] = "CUDA is great. CUDA is fast. CUDA is powerful.";
    char word[] = "CUDA";
    int count = 0;

    char *d_sentence, *d_word;
    int *d_count;

    hipMalloc((void **)&d_sentence, sizeof(sentence));
    hipMalloc((void **)&d_word, sizeof(word));
    hipMalloc((void **)&d_count, sizeof(int));

    hipMemcpy(d_sentence, sentence, sizeof(sentence), hipMemcpyHostToDevice);
    hipMemcpy(d_word, word, sizeof(word), hipMemcpyHostToDevice);
    hipMemcpy(d_count, &count, sizeof(int), hipMemcpyHostToDevice);

    countWordOccurrences<<<1, strlen(sentence)>>>(d_sentence, d_word, d_count);

    hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);

    printf("The word '%s' appears %d times.\n", word, count);

    hipFree(d_sentence);
    hipFree(d_word);
    hipFree(d_count);

    return 0;
}
