#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void addVectors(float *A, float *B, float *C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) C[i] = A[i] + B[i];
}

int main() {
    int N = 5;
    float A[N] = {1, 2, 3, 4, 5}, B[N] = {10, 20, 30, 40, 50}, C[N];
    float *d_A, *d_B, *d_C;
    
    hipMalloc(&d_A, N * sizeof(float));
    hipMalloc(&d_B, N * sizeof(float));
    hipMalloc(&d_C, N * sizeof(float));
    
    hipMemcpy(d_A, A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * sizeof(float), hipMemcpyHostToDevice);

    addVectors<<<1, N>>>(d_A, d_B, d_C, N);
    
    hipMemcpy(C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);
    
    for (int i = 0; i < N; i++)
        printf("%f ", C[i]);
    
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    return 0;
}