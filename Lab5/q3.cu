#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void computeSine(float *angles, float *results, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) results[i] = sinf(angles[i]);
}

int main() {
    int N = 5;
    float angles[N] = {0, 1.57, 3.14, 4.71, 6.28}, results[N];
    float *d_angles, *d_results;
    
    hipMalloc(&d_angles, N * sizeof(float));
    hipMalloc(&d_results, N * sizeof(float));
    
    hipMemcpy(d_angles, angles, N * sizeof(float), hipMemcpyHostToDevice);

    computeSine<<<1, N>>>(d_angles, d_results, N);
    
    hipMemcpy(results, d_results, N * sizeof(float), hipMemcpyDeviceToHost);
    
    for (int i = 0; i < N; i++)
        printf("%f ", results[i]);

    hipFree(d_angles); hipFree(d_results);
    return 0;
}