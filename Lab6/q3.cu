#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void oddEvenSort(int *a, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    for (int phase = 0; phase < n; phase++) {
        if ((phase % 2 == 0 && tid % 2 == 0 && tid < n - 1) || 
            (phase % 2 == 1 && tid % 2 == 1 && tid < n - 1)) {
            if (a[tid] > a[tid + 1]) {
                int temp = a[tid];
                a[tid] = a[tid + 1];
                a[tid + 1] = temp;
            }
        }
        __syncthreads();
    }
}

int main() {
    int n = 8;
    int h_a[] = {7, 3, 8, 6, 2, 5, 4, 1};

    int *d_a;
    hipMalloc((void **)&d_a, n * sizeof(int));
    hipMemcpy(d_a, h_a, n * sizeof(int), hipMemcpyHostToDevice);

    oddEvenSort<<<1,n>>>(d_a,n);

    hipMemcpy(h_a,d_a,n*sizeof(int),hipMemcpyDeviceToHost);

   printf("Sorted Array:\n");
   for(int i=0;i<n;++i){
      printf("%d ",h_a[i]);
   }

   hipFree(d_a);
   return(0);
}
